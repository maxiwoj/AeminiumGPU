
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// #define START_LENGTH 134217728
#define START_LENGTH 16384
// #define START_LENGTH 8192
// #define START_LENGTH 4096
// #define START_LENGTH 2048
// #define START_LENGTH 100
#define MAX_BLOCK_SIZE 1024

// universal
int predicate(int value) {
    if( (value > 7) && (value % 2) && !(value % 3) ) {
        return 1;
    } else {
        return 0;
    }
}

// cpu
int* check_array(int *array, int length) {
    int *array_out = (int*)malloc(sizeof(int) * length);
    for(int i=0; i<length; i++) {
        array_out[i] = predicate(array[i]);
    }
    return array_out;
}

// cpu
int* prefix_sum(int *array, int length) {
    int *array_out = (int*)malloc(sizeof(int) * length);
    array_out[0] = array[0];
    for(int i=1; i<length; i++) {
        array_out[i] = array[i] + array_out[i-1];
    }
    return array_out;
}

// cpu
int* sieve_out(int *values, int *booleans, int *indexes, int *length) {
    int new_length = 0;
    for(int i=0; i<*length; i++) {
        if(booleans[i]) {
            new_length++;
        }
    }

    int *array_out = (int*)malloc(sizeof(int) * new_length);
    for(int i=0; i<*length; i++) {
        if(booleans[i]) {
            array_out[indexes[i]-1] = values[i];
        }
    }

    *length = new_length;
    return array_out;
}

void print_array(int *array, int length) {
    int *ptr = array;
    int counter = 0;
    while(ptr < array+length) {
        printf("array[%d] = %d\n", counter, *ptr);
        counter++;
        ptr++;
    }
}

// cpu
int* seq_filter(int *array, int *length) {
    int *checked = check_array(array, *length);
    int *aggregated = prefix_sum(checked, *length);
    int *filtered = sieve_out(array, checked, aggregated, length);

    return filtered;
}

// #pragma OPENCL EXTENSION cl_khr_global_int32_base_atomics : enable

__global__ void filterKernel(int *values, int *bools, int *prefix_sum, int *output, int *array_length, int *semaphore) {
    int threads = *array_length / 2 + *array_length % 2;
    int blocks = threads / MAX_BLOCK_SIZE + (threads % MAX_BLOCK_SIZE > 0);
    int threads_per_block = (threads + blocks-1) / blocks;
    int idx = 2*threadIdx.x + (1 - threads%2) + (threads % blocks > 0);
    int modulo = 2;

    for(int i=0; i<threads; i+=threads_per_block) {
        if(i+idx < *array_length) {
            if((values[i+idx] > 7) && (values[i+idx] % 2) && !(values[i+idx] % 3)) {
                bools[i+idx] = 1;
                prefix_sum[i+idx] = 1;
            } else {
                bools[i+idx] = 0;
                prefix_sum[i+idx] = 0;
            }

            if(i+idx > 0) {
                if((values[i+idx-1] > 7) && (values[i+idx-1] % 2) && !(values[i+idx-1] % 3)) {
                    bools[i+idx-1] = 1;
                    prefix_sum[i+idx-1] = 1;
                } else {
                    bools[i+idx-1] = 0;
                    prefix_sum[i+idx-1] = 0;
                }
            }
        }
    }

    __syncthreads();  // <---------<< 

    while(modulo < *array_length) {
        for(int i=0; i<threads; i+=threads_per_block) {
            if((i+idx % modulo) == ((*array_length-1) % modulo)) {
                if((i+idx - modulo/2) >= 0) {
                    prefix_sum[i+idx] += prefix_sum[i+idx-modulo/2];
                }
            }
        }
        __syncthreads();  // <---------<< 
        modulo *= 2;
    }

    if(threadIdx.x == 0) {  // <---------<< 
        prefix_sum[*array_length-1] = 0;
    }

    while(modulo >= 2) {
        for(int i=0; i<threads; i+=threads_per_block) {
            if((i+idx % modulo) == ((*array_length-1) % modulo)) {
                if((i+idx - modulo/2) >= 0) {
                    int temp = prefix_sum[i+idx];
                    prefix_sum[i+idx] += prefix_sum[i+idx - modulo/2];
                    prefix_sum[i+idx - modulo/2] = temp;
                }
            }
        }
        __syncthreads();  // <---------<< 
        modulo /= 2;
    }

    for(int i=0; i<threads; i+=threads_per_block) {
        if(bools[i+idx]) {
            prefix_sum[i+idx] += 1;
            output[prefix_sum[i+idx]-1] = values[i+idx];
        }
        if(((i+idx - 1) >= 0) && (bools[i+idx-1])) {
            prefix_sum[i+idx-1] += 1;
            output[prefix_sum[i+idx-1]-1] = values[i+idx-1];
        }
        __syncthreads();  // <---------<< 
    }

    if(threadIdx.x == 0) {  // <---------<< 
        *array_length = prefix_sum[*array_length-1];
    }

}

int* gpu_filter(int* array, int *length) {
    int *values, *bools, *prefix_sum, *output, *array_length, *semaphore;
    int size = sizeof(int) * *length;
    hipMalloc(&values, size);  // <---------<< 
    hipMalloc(&bools, size);  // <---------<< 
    hipMalloc(&prefix_sum, size);  // <---------<< 
    hipMalloc(&output, size);  // <---------<< 
    hipMalloc(&array_length, sizeof(int));  // <---------<< 
    hipMalloc(&semaphore, sizeof(int)*2);  // <---------<< 
    hipMemcpy(values, array, size, hipMemcpyHostToDevice);  // <---------<< 
    hipMemcpy(array_length, length, sizeof(int), hipMemcpyHostToDevice);  // <---------<< 

    
    int threads = *length / 2 + *length % 2;
    int blocks = threads / MAX_BLOCK_SIZE + (threads % MAX_BLOCK_SIZE > 0);
    int threads_per_block = (threads + blocks-1) / blocks;
    // printf("length:%d -> threads:%d -> blocks:%d -> threads_per_block:%d\n", *length, threads, blocks, threads_per_block);

    filterKernel<<<1, threads_per_block>>>(values, bools, prefix_sum, output, array_length, semaphore);  // <---------<< 

    hipMemcpy(length, array_length, sizeof(int), hipMemcpyDeviceToHost);  // <---------<< 
    int new_size = *length * sizeof(int);  // <---------<< 
    int *h_output = (int*)malloc(new_size);
    hipMemcpy(h_output, output, new_size, hipMemcpyDeviceToHost);  // <---------<< 
    hipFree(&values);  // <---------<< 
    hipFree(&bools);  // <---------<< 
    hipFree(&prefix_sum);  // <---------<< 
    hipFree(&output);  // <---------<< 
    hipFree(&array_length);  // <---------<< 
    hipFree(&semaphore);  // <---------<< 

    return h_output;
}

int main() {
    srand(1920);
    int length = START_LENGTH;
    int repeats = 10000;

    int *array = (int*)malloc(sizeof(int) * length);
    for(int i=0, *ptr=array; i<length; i++, ptr++) {
        *ptr = rand()%length;
    }

    int *filtered = seq_filter(array, &length);
    // print_array(filtered, length);
    printf("\x1b[1;40;34mCPU done\x1b[0m\n");

    // printf("\n-------------------- array[0] = %d --------------------\n\n", array[0]);

    int i = 0;
    bool broken = false;

    while(!broken && (i < repeats)) {
        length = START_LENGTH;
        int* gpu_filtered = gpu_filter(array, &length);
        // print_array(gpu_filtered, length);
    
        // check
        bool coherent = true;
        int iter = 0;
        while(coherent && (iter < length)) {
            if(filtered[iter] != gpu_filtered[iter]) {
                coherent = false;
            }
            iter++;
        }
        // if(coherent)
        //     printf("\x1b[1;40;32mCorrect :)\x1b[0m\n");
        // else {
        if(!coherent) {
            broken = true;
            printf("\x1b[1;40;31mIncorrect!\x1b[0m\npassed correctly %d times\n", i);
        }
        i++;
    }
    if(i == repeats) {
        printf("\x1b[1;40;33mSuccess!\x1b[0m\n");
    }
    
}